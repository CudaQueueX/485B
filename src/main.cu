#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "BPQ.h"

// k := the max batch size per "node" in the priority queue. This batch size is equivalent |cols|

#define rows 3
#define cols 3

int main() {
    

    BPQ test(3,3);

    int items1[3] = {1, 1, 1};  // Example row of items to insert
    int items2[3] = {2,2,2};
    int items3[3] = {3,3,3};
    test.insert(items1, 0);        // Insert into row 1
    test.insert(items2, 1);
    test.insert(items3, 2);

    test.test_print("1D");        // Print the updated array  

    test.swapRows(0,2);           // Swap rows 1 and 2

    test.test_print("1D");

    // test.test_insert();
    // std::cout << std::endl;
    // test.test_print("2D");
    // std::cout << std::endl;

    // // test.flatten();
    // test.test_print("1D");
    // std::cout << std::endl;

    // // std::cout << "Transferring to device" << std::endl;
    // test.transfer_to_device();
    
    // // std::cout << "Testing add" << std::endl;
    // test.test_add(1);

    // test.swapRows(0,1);
    

    
    return 0;
}





