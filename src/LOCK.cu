#include "hip/hip_runtime.h"


#include "LOCK.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>


__device__ void LOCK::lock() {
    while (atomicCAS(&mutex, 0, 1) != 0);
}

__device__ void LOCK::unlock() {
    atomicExch(&mutex, 0);
}